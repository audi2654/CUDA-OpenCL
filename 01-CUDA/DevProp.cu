//26/03/2022
//HPP CUDA Program to show CUDA GPU Device Properties

//cmd : nvcc DevProp.cu -o DevProp

#include <stdio.h>

int main(void)
{
    //function prototype
    void PrintCUDADeviceProperties(void);

    //code
    PrintCUDADeviceProperties();
}

void PrintCUDADeviceProperties(void)
{
    //code
    printf("CUDA INFORMATION\n");
    printf("=============================================================================================\n");
    hipError_t ret_cuda_rt;
    int dev_count;

    ret_cuda_rt = hipGetDeviceCount(&dev_count);
    if(ret_cuda_rt != hipSuccess)
    {
        printf("CUDA Runtime API Error - hipGetDeviceCount() failed due to %s\n", hipGetErrorString(ret_cuda_rt));
        printf("In file : %s on line : %d\n", __FILE__,  __LINE__);    //shows line & file of error
    }
    else if(dev_count == 0)
    {
        printf("There is no CUDA supported device on this system\n");
        return;
    }
    else
    {
        printf("Total number of CUDA supported GPU Devices on this system : %d\n", dev_count);
        
        for(int i = 0; i < dev_count; i++)
        {
            hipDeviceProp_t dev_prop;
            int driverVersion = 0, runtimeVersion = 0;

            ret_cuda_rt = hipGetDeviceProperties(&dev_prop, i);
            if(ret_cuda_rt != hipSuccess)
            {
                printf("%s in %s at line %d\n", hipGetErrorString(ret_cuda_rt), __FILE__, __LINE__);
                return;
            }

            hipDriverGetVersion(&driverVersion);
            hipRuntimeGetVersion(&runtimeVersion);
            printf("\n");
            printf("******* CUDA DRIVER & RUNTIME INFORMATION *******\n");
            printf("=============================================================================================\n");
            printf("CUDA Driver Version                                 : %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10);
            printf("CUDA Runtime Version                                : %d.%d\n", runtimeVersion / 1000, (runtimeVersion % 100) / 10);
            
            printf("\n");
            printf("=============================================================================================\n");
            printf("******* GPU DEVICE GENERAL INFORMATION ********\n");
            printf("=============================================================================================\n");
            printf("GPU Device Number                                   : %d\n", i);
            printf("GPU Device Name                                     : %s\n", dev_prop.name);
            printf("GPU Device Compute Compatibility                    : %d.%d\n", dev_prop.major, dev_prop.minor);
            printf("GPU Device Clock Rate                               : %d\n", dev_prop.clockRate);
            
            printf("GPU Device Type                                     : ");
            if(dev_prop.integrated)
            {
                printf("Integrated (On-Board)\n");
            }
            else
            {
                printf("Discrete (Separate Card)\n");
            }

            printf("\n");
            printf("******* GPU DEVICE MEMORY INFORMATION ********\n");
            printf("=============================================================================================\n");
            printf("GPU Device Total Memory                             : %.0f GB = %0.f MB = %llu Bytes\n", 
                                                                        ((float)dev_prop.totalGlobalMem / 1048576.0f) / 1024.0f,
                                                                        (float)dev_prop.totalGlobalMem / 1048576.0f,
                                                                        (unsigned long long)dev_prop.totalGlobalMem);
            printf("GPU Device Constant Memory                          : %lu Bytes\n", (unsigned long)dev_prop.totalConstMem);
            printf("GPU Device Shared Memory per SMProcesser            : %d\n", (unsigned long)dev_prop.sharedMemPerBlock);

            printf("\n");
            printf("******* GPU DEVICE MULTIPROCESSOR INFORMATION ********\n");
            printf("=============================================================================================\n");
            printf("GPU Device No. of SMProcessors                      : %d\n", dev_prop.multiProcessorCount);
            printf("GPU Device No. of Registers Per SMProcessor         : %d\n", dev_prop.regsPerBlock);

            printf("\n");
            printf("******* GPU DEVICE THREAD INFORMATION ********\n");
            printf("=============================================================================================\n");
            printf("GPU Device Max No. of Threads Per SMProcessor       : %d\n", dev_prop.maxThreadsPerMultiProcessor);
            printf("GPU Device Max No. of Threads Per Block             : %d\n", dev_prop.maxThreadsPerBlock);
            printf("GPU Device Threads in Warp                          : %d\n", dev_prop.warpSize);
            printf("GPU Device Max Thread Dimensions                    : %d, %d, %d\n", dev_prop.maxThreadsDim[0], dev_prop.maxThreadsDim[1], dev_prop.maxThreadsDim[2]);
            printf("GPU Device Max Grid Dimensions                      : %d, %d, %d\n", dev_prop.maxGridSize[0], dev_prop.maxGridSize[1], dev_prop.maxGridSize[2]);
            
            printf("\n");
            printf("******* GPU DEVICE DRIVER INFORMATION ********\n");
            printf("=============================================================================================\n");
            printf("GPU Device has ECC Support                          : %s\n", dev_prop.ECCEnabled ? "Enabled" : "Disabled");

            #if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
                printf("GPU Device CUDA Driver Mode (TCC or WDDM)           : %s\n", dev_prop.tccDriver ? "TCC (Tesla Compute Cluster)" : "WDDM (Windows Display Driver Model)");
            #endif

            printf("=============================================================================================\n");
        }
    }
}
